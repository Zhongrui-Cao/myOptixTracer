#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Config buffer
rtBuffer<Config> config;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{
    Config cf = config[0];

    MaterialProperties mv = attrib.phongmat;

    float3 result = make_float3(0, 0, 0);

    if (attrib.isQuadLight) {
        payload.radiance = mv.emission;
        return;
    }

    float3 r = normalize(reflect(-attrib.wo, attrib.normal));

    for (int i = 0; i < qlights.size(); i++)
    {
        QuadLight ql = qlights[i];
        float3 sum = make_float3(0, 0, 0);
        // random sampling
        for (int i = 0; i < cf.lightSamples; i++) {
            float u1 = rnd(payload.seed);
            float u2 = rnd(payload.seed);
            float3 xprime = ql.a + u1 * ql.ab + u2 * ql.ac;
            float3 wi = normalize(xprime - attrib.intersection);

            float3 brdf_diffuse = (mv.diffuse / M_PIf);
            float rdotWiPows = powf(clamp(dot(r, wi), 0.0f, M_PIf / 2.0f), mv.shininess);
            float3 brdf_specular = mv.specular * ((mv.shininess + 2) / (2 * M_PIf)) * rdotWiPows;
            float3 brdf = brdf_diffuse + brdf_specular;

            float g1 = clamp(dot(attrib.normal, wi), 0.0f, M_PIf / 2.0f);
            float3 nl = normalize(cross(ql.ab, ql.ac));
            float g2 = clamp(dot(nl, wi), 0.0f, M_PIf / 2.0f);
            float geometryTerm = (g1 * g2) / powf(length(attrib.intersection - xprime), 2.0f);
            
            sum += brdf * geometryTerm * 1;
        }

        float A = length(cross(ql.ab, ql.ac));

        result += ql.intensity * (A / cf.lightSamples) * sum;
    }

    // Compute the final radiance
    payload.radiance = result;

    payload.done = true;

}