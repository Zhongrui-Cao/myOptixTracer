#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Config buffer
rtBuffer<Config> config;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{
    Config cf = config[0];

    MaterialProperties mv = attrib.phongmat;

    float3 result = make_float3(0, 0, 0);

    if (attrib.isQuadLight) {
        payload.radiance = mv.emission;
        return;
    }

    float3 r = normalize(reflect(-attrib.wo, attrib.normal));

    for (int i = 0; i < qlights.size(); i++)
    {
        QuadLight ql = qlights[i];
        float3 sum = make_float3(0, 0, 0);
        
        for (int i = 0; i < cf.lightSamples; i++) {
            //for tratified sampling
            int M = sqrt((double)cf.lightSamples);
            int row = i / M;
            int col = i % M;

            float u1 = rnd(payload.seed);
            float u2 = rnd(payload.seed);
            //light sample position
            float3 xprime = ql.a + u1 * ql.ab + u2 * ql.ac;
            if (cf.lightStratify == true) {
                xprime = ql.a + (col + u1) / M * ql.ab + (row + u2) / M * ql.ac;
            }
            //light sample direction
            float3 wi = normalize(xprime - attrib.intersection);

            //calc brdf
            float3 brdf_diffuse = (mv.diffuse / M_PIf);
            float rdotWiPows = powf(clamp(dot(r, wi), 0.0f, M_PIf / 2.0f), mv.shininess);

            float3 brdf_specular = make_float3(0, 0, 0);
            if (length(mv.specular) > 0) {
                brdf_specular = mv.specular * ((mv.shininess + 2) / (2 * M_PIf)) * rdotWiPows;
            }
            
            float3 brdf = brdf_diffuse + brdf_specular;
            //rtPrintf("specular term is: %f, %f, %f\n", brdf_specular.x, brdf_specular.y, brdf_specular.z);

            //calc geometry term
            float g1 = clamp(dot(attrib.normal, wi), 0.0f, M_PIf / 2.0f);
            float3 nl = normalize(cross(ql.ab, ql.ac));
            float g2 = clamp(dot(nl, wi), 0.0f, M_PIf / 2.0f);
            float geometryTerm = (g1 * g2) / powf(length(attrib.intersection - xprime), 2.0f);

            //shoot shadow ray
            float lightDist = length(xprime - attrib.intersection);
            ShadowPayload shadowPayload;
            shadowPayload.isVisible = true;
            Ray shadowRay = make_Ray(attrib.intersection + wi * 0.001f,
                wi, 1, 0.001f, lightDist - 0.1f);
            rtTrace(root, shadowRay, shadowPayload);

            float visibility;
            if (shadowPayload.isVisible) {
                visibility = 1.0f;
            }
            else {
                visibility = 0.0f;
            }
            
            sum += brdf * geometryTerm * visibility;
        }

        float A = length(cross(ql.ab, ql.ac));

        result += ql.intensity * (A / cf.lightSamples) * sum;
    }

    // Compute the final radiance
    payload.radiance = result * payload.throughput;

    payload.done = true;

}