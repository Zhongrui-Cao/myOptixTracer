#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    float t;

    // TODO: implement sphere intersection test here
    float3 p0 = make_float3(sphere.transform.inverse() * make_float4(ray.origin, 1) + 0.01f * make_float4(ray.direction, 0));
    float3 d  = make_float3(normalize(sphere.transform.inverse() * make_float4(ray.direction, 0)));

    float3 oc   = p0 - sphere.center;
    float a     = dot(d, d);
    float halfb = dot(d, oc);
    float b     = 2.0f * halfb;
    float c     = dot(oc, oc) - sphere.radius * sphere.radius;

    float discriminant = b * b - 4 * a * c;

    // roots
    float rootminus = (-b - sqrt(discriminant)) / (2.f * a);
    float rootplus  = (-b + sqrt(discriminant)) / (2.f * a);

    // complex root
    if (discriminant < 0) {
        return;
    }
    else if (discriminant > 0.f) {
        //choose positive one if 2 real roots
        if (((rootminus > 0) && (rootplus < 0)) || ((rootminus < 0) && (rootplus > 0))) {
            t = (rootminus > 0 ? rootminus : rootplus);
        }
        //choose the smaller one if 2 positive roots
        else if (rootminus > 0 && rootplus > 0) {
            t = (rootminus > rootplus ? rootplus : rootminus);
        }
        else {
            return; 
        }
    }
    else {
        // one root
        t = rootminus;
    }


    float3 p = (p0 + d * t);
    float4 intercection = sphere.transform * make_float4(p, 1);
    p = make_float3(intercection) / intercection.w; // intersection in the world space
    t = length(p - ray.origin);

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib.phongmat = sphere.phongmat;
        attrib.intersection = p;
        attrib.wo = -ray.direction;
        float4 tintersection = sphere.transform.inverse() * make_float4(attrib.intersection, 1);
        attrib.normal = normalize(make_float3(tintersection) / tintersection.w);
        attrib.normal = normalize(make_float3(sphere.transform.inverse().transpose() * make_float4(attrib.normal, 0)));
        attrib.isQuadLight = false;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // TODO: implement sphere bouding box
    float x, y, z;
    x = length(make_float3(sphere.transform.getRow(0)));
    y = length(make_float3(sphere.transform.getRow(1)));
    z = length(make_float3(sphere.transform.getRow(2)));
    result[0] = sphere.transform[3] - x;
    result[1] = sphere.transform[7] - y;
    result[2] = sphere.transform[11] - z;
    result[3] = sphere.transform[3] + x;
    result[4] = sphere.transform[7] + y;
    result[5] = sphere.transform[11] + z;
}