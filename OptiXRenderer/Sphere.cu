#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    Matrix4x4 itrans = sphere.transform.inverse();
    float4 rayOriH = itrans * make_float4(ray.origin, 1);
    float3 rayOri = make_float3(rayOriH) / rayOriH.w;
    float3 rayDir = normalize(make_float3(itrans * make_float4(ray.direction, 0)));

    float t = 0;
    float3 CP0 = rayOri;
    float P1dotCP0 = dot(rayDir, CP0);
    float CP0dotCP0 = dot(CP0, CP0);
    float r2 = 1.f;
    float disc = P1dotCP0 * P1dotCP0 - CP0dotCP0 + r2;
    if (disc < 0) return;
    if (disc == 0)
        t = -P1dotCP0;
    else if (CP0dotCP0 > r2)
        t = -P1dotCP0 - sqrt(disc);
    else
        t = -P1dotCP0 + sqrt(disc);

    if (t < 0.01f) return;

    // Intersection is found
    float3 P = rayOri + t * rayDir; // intersection in the object space
    float4 intersectionH = sphere.transform * make_float4(P, 1);
    P = make_float3(intersectionH) / intersectionH.w; // intersection in the world space
    t = length(P - ray.origin); // distance

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib.phongmat = sphere.phongmat;
        attrib.intersection = P;
        attrib.wo = -ray.direction;
        float4 tintersection = sphere.transform.inverse() * make_float4(attrib.intersection, 1);
        attrib.normal = normalize(make_float3(tintersection) / tintersection.w);
        attrib.normal = normalize(make_float3(sphere.transform.inverse().transpose() * make_float4(attrib.normal, 0)));
        attrib.isQuadLight = false;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // TODO: implement sphere bouding box
    float x, y, z;
    x = length(make_float3(sphere.transform.getRow(0)));
    y = length(make_float3(sphere.transform.getRow(1)));
    z = length(make_float3(sphere.transform.getRow(2)));
    result[0] = sphere.transform[3] - x;
    result[1] = sphere.transform[7] - y;
    result[2] = sphere.transform[11] - z;
    result[3] = sphere.transform[3] + x;
    result[4] = sphere.transform[7] + y;
    result[5] = sphere.transform[11] + z;
}