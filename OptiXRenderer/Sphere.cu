#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    float t;

    // TODO: implement sphere intersection test here
    float3 p0 = make_float3(sphere.transform.inverse() * make_float4(ray.origin, 1) + ray.tmin * make_float4(ray.direction, 0));
    float3 d  = make_float3(normalize(sphere.transform.inverse() * make_float4(ray.direction, 0)));

    float3 oc   = p0 - sphere.center;
    float a     = dot(d, d);
    float halfb = dot(d, oc);
    float b     = 2.0f * halfb;
    float c     = dot(oc, oc) - sphere.radius * sphere.radius;

    float discriminant = b * b - 4 * a * c;

    // roots
    float rootminus = (-b - sqrt(discriminant)) / (2.f * a);
    float rootplus  = (-b + sqrt(discriminant)) / (2.f * a);

    // complex root
    if (discriminant < 0) {
        return;
    }
    else if (discriminant > 0.f) {
        //choose positive one if 2 real roots
        if (((rootminus > 0) && (rootplus < 0)) || ((rootminus < 0) && (rootplus > 0))) {
            t = (rootminus > 0 ? rootminus : rootplus);
        }
        //choose the smaller one if 2 positive roots
        else if (rootminus > 0 && rootplus > 0) {
            t = (rootminus > rootplus ? rootplus : rootminus);
        }
        else {
            return; 
        }
    }
    else {
        // one root
        t = rootminus;
    }

    float3 p = (p0 + d * t);
    //TODO
    float3 intercection = make_float3(sphere.transform * make_float4(p, 1));
    t = length(intercection - ray.origin);

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib = sphere.attribute;
        // TODO: assign attribute variables here

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // TODO: implement sphere bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}