#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Config buffer
rtBuffer<Config> config;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{
    Config cf = config[0];

    MaterialProperties mv = attrib.phongmat;

    float3 result = make_float3(0, 0, 0);

    if (attrib.isQuadLight) {
        if (!cf.nextEventEstimation || payload.depth == 0) {
            payload.radiance += mv.emission * payload.throughput;
            payload.done = true;
            return;
        }
        payload.done = true;
        return;
    }

    float3 r = normalize(reflect(-attrib.wo, attrib.normal));
    // direct lighting
    for (int i = 0; i < qlights.size(); i++)
    {
        QuadLight ql = qlights[i];
        float3 sum = make_float3(0, 0, 0);

        float u1 = rnd(payload.seed);
        float u2 = rnd(payload.seed);
        //light sample position
        float3 xprime = ql.a + u1 * ql.ab + u2 * ql.ac;

        //light sample direction
        float3 wi = normalize(xprime - attrib.intersection);

        //calc brdf
        float3 brdf_diffuse = (mv.diffuse / M_PIf);
        float rdotWiPows = powf(clamp(dot(r, wi), 0.0f, M_PIf / 2.0f), mv.shininess);
        float3 brdf_specular = make_float3(0, 0, 0);
        if (length(mv.specular) > 0) {
            brdf_specular = mv.specular * ((mv.shininess + 2) / (2 * M_PIf)) * rdotWiPows;
        }
        float3 brdf = brdf_diffuse + brdf_specular;

        //calc geometry term
        float g1 = clamp(dot(attrib.normal, wi), 0.0f, M_PIf / 2.0f);
        float3 nl = normalize(cross(ql.ab, ql.ac));
        float g2 = clamp(dot(nl, wi), 0.0f, M_PIf / 2.0f);
        float geometryTerm = (g1 * g2) / powf(length(attrib.intersection - xprime), 2.0f);

        //shoot shadow ray
        float lightDist = length(xprime - attrib.intersection);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        // lightDist - 0.01f to not let the trangles shadow the entire light
        Ray shadowRay = make_Ray(attrib.intersection + wi * 0.001f,
            wi, 1, 0.001f, lightDist - 0.01f);
        rtTrace(root, shadowRay, shadowPayload);

        float visibility;
        if (shadowPayload.isVisible) {
            visibility = 1.0f;
        }
        else {
            visibility = 0.0f;
        }

        sum = brdf * geometryTerm * visibility;

        float A = length(cross(ql.ab, ql.ac));

        result += ql.intensity * A * sum;
    }

    if (cf.nextEventEstimation) {
        payload.radiance += result * payload.throughput;
        if (payload.depth >= cf.maxDepth - 1 && !cf.russianRoulette) {
            payload.done = true;
            return;
        }
    }

    // russian roulette
    if (cf.russianRoulette) {
        // load gun
        float q = 1.0f - fminf(fmaxf(fmaxf(payload.throughput.x, payload.throughput.y), payload.throughput.z), 1.0f);
        // spin wheel
        //unsigned int world = tea<16>(rnd(payload.seed), rnd(payload.seed));
        float spin = rnd(payload.seed);
        // pull trigger
        if (spin < q) {
            // get killed, no indirect ray shot
            payload.done = true;
            return;
        }
        else {
            // alive with boosted throughput
            payload.throughput *= 1.0f / (1.0f - q);
        }
    }

    // indirect lighting
    float xi1 = rnd(payload.seed);
    float xi2 = rnd(payload.seed);

    float theta = acosf(xi1);
    float phi = 2.0f * M_PIf * xi2;

    float3 s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));

    float3 w = normalize(attrib.normal);
    float3 up = make_float3(0, 1, 0);
    //float3 a = length(normalize(w - up)) < 0.1f ? make_float3(1, 0, 0) : up;
    //TODO if a close to w what to do
    float3 a = make_float3(1, 2, 3);
    float3 u = normalize(cross(a, w));
    float3 v = normalize(cross(w, u));
    float3 wi = s.x * u + s.y * v + s.z * w;

    //calc brdf
    float3 brdf_diffuse = (mv.diffuse / M_PIf);
    float rdotWiPows = powf(clamp(dot(r, wi), 0.0f, M_PIf / 2.0f), mv.shininess);
    float3 brdf_specular = make_float3(0, 0, 0);
    if (length(mv.specular) > 0) {
        brdf_specular = mv.specular * ((mv.shininess + 2) / (2 * M_PIf)) * rdotWiPows;
    }
    float3 brdf = brdf_diffuse + brdf_specular;
    
    // Set origin and dir for tracing the reflection ray
    payload.origin = attrib.intersection;
    payload.dir = wi; // random reflection
    payload.depth++;
    payload.throughput *= 2.0f * M_PIf * brdf * dot(attrib.normal, wi);

}