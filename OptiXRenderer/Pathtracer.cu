#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Config buffer
rtBuffer<Config> config;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{
    Config cf = config[0];

    MaterialProperties mv = attrib.phongmat;

    float3 result = make_float3(0, 0, 0);

    if (attrib.isQuadLight) {
        payload.radiance = mv.emission * payload.throughput;
        payload.done = true;
        return;
    }

    float3 r = normalize(reflect(-attrib.wo, attrib.normal));
        
    float xi1 = rnd(payload.seed);
    float xi2 = rnd(payload.seed);

    float theta = acosf(xi1);
    float phi   = 2.0f * M_PIf * xi2;

    float3 s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));

    float3 w = normalize(attrib.normal);
    float3 up = make_float3(0, 1, 0);
    //float3 a = length(normalize(w - up)) < 0.1f ? make_float3(1, 0, 0) : up;
    //TODO if a close to w what to do
    float3 a = make_float3(1, 2, 3);
    float3 u = normalize(cross(a, w));
    float3 v = normalize(cross(w, u));
    float3 wi = s.x * u + s.y * v + s.z * w;

    //calc brdf
    float3 brdf_diffuse = (mv.diffuse / M_PIf);
    float rdotWiPows = powf(clamp(dot(r, wi), 0.0f, M_PIf / 2.0f), mv.shininess);
    float3 brdf_specular = make_float3(0, 0, 0);
    if (length(mv.specular) > 0) {
        brdf_specular = mv.specular * ((mv.shininess + 2) / (2 * M_PIf)) * rdotWiPows;
    }
    float3 brdf = brdf_diffuse + brdf_specular;

    // Set origin and dir for tracing the reflection ray
    payload.origin = attrib.intersection;
    payload.dir = wi; // random reflection
    payload.depth++;
    payload.throughput *= 2.0f * M_PIf * brdf * dot(attrib.normal, wi);

}