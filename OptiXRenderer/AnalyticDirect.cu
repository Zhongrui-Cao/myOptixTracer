#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{
    MaterialProperties mv = attrib.phongmat;

    float3 result = mv.emission;

    float3 bdrf = mv.diffuse / 3.14159265358979323846f;

    if (attrib.isQuadLight) {
        payload.radiance = result;
        return;
    }

    for (int i = 0; i < qlights.size(); i++)
    {
        float3 a = qlights[i].a;
        float3 b = qlights[i].a + qlights[i].ab;
        float3 d = b + qlights[i].ac;
        float3 c = a + qlights[i].ac;

        float thetaK = acosf(dot(normalize(a - attrib.intersection), normalize(b - attrib.intersection)));
        float3 gammaK = normalize(cross((a - attrib.intersection), (b - attrib.intersection)));
        float3 tg = thetaK * gammaK;

        float thetaK1 = acosf(dot(normalize(b - attrib.intersection), normalize(d - attrib.intersection)));
        float3 gammaK1 = normalize(cross((b - attrib.intersection), (d - attrib.intersection)));
        float3 tg1 = thetaK1 * gammaK1;

        float thetaK2 = acosf(dot(normalize(d - attrib.intersection), normalize(c - attrib.intersection)));
        float3 gammaK2 = normalize(cross((d - attrib.intersection), (c - attrib.intersection)));
        float3 tg2 = thetaK2 * gammaK2;

        float thetaK3 = acosf(dot(normalize(c - attrib.intersection), normalize(a - attrib.intersection)));
        float3 gammaK3 = normalize(cross((c - attrib.intersection), (a - attrib.intersection)));
        float3 tg3 = thetaK3 * gammaK3;

        float3 Phi = (tg + tg1 + tg2 + tg3) / 2.0f;

        result += bdrf * qlights[i].intensity * dot(Phi, attrib.normal);
    }

    // Calculate the direct illumination of point lights
    for (int i = 0; i < plights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = normalize(plights[i].location - attrib.intersection);
        float lightDist = length(plights[i].location - attrib.intersection);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * 0.001f,
            lightDir, 1, 0.001f, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float att = dot(plights[i].attenuation, make_float3(1, lightDist, lightDist * lightDist));
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= plights[i].color / att;
            result += I;
        }
    }

    // Calculate the direct illumination of directional lights
    for (int i = 0; i < dlights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = dlights[i].direction;
        float lightDist = RT_DEFAULT_MAX;
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * 0.001f,
            lightDir, 1, 0.001f, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= dlights[i].color;
            result += I;
        }
    }

    // Compute the final radiance
    payload.radiance = result * payload.throughput;

    // Calculate reflection
    if (length(mv.specular) > 0)
    {
        // Set origin and dir for tracing the reflection ray
        payload.origin = attrib.intersection;
        payload.dir = reflect(-attrib.wo, attrib.normal); // mirror reflection

        payload.depth++;
        payload.throughput *= mv.specular;
    }
    else
    {
        payload.done = true;
    }
}