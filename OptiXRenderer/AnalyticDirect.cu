#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{
    float PI_F = 3.1415926535f;
    MaterialProperties mv = attrib.phongmat;

    float3 result = mv.emission;

    float3 brdf = (mv.diffuse / PI_F);

    if (attrib.isQuadLight) {
        payload.radiance = result;
        return;
    }

    for (int i = 0; i < qlights.size(); i++)
    {
        float3 a = qlights[i].a;
        float3 b = qlights[i].a + qlights[i].ab;
        float3 d = b + qlights[i].ac;
        float3 c = a + qlights[i].ac;

        float thetaK = acosf(dot(normalize(a - attrib.intersection), normalize(b - attrib.intersection)));
        float3 gammaK = normalize(cross((a - attrib.intersection), (b - attrib.intersection)));
        float3 tg = thetaK * gammaK;

        float thetaK1 = acosf(dot(normalize(b - attrib.intersection), normalize(d - attrib.intersection)));
        float3 gammaK1 = normalize(cross((b - attrib.intersection), (d - attrib.intersection)));
        float3 tg1 = thetaK1 * gammaK1;

        float thetaK2 = acosf(dot(normalize(d - attrib.intersection), normalize(c - attrib.intersection)));
        float3 gammaK2 = normalize(cross((d - attrib.intersection), (c - attrib.intersection)));
        float3 tg2 = thetaK2 * gammaK2;

        float thetaK3 = acosf(dot(normalize(c - attrib.intersection), normalize(a - attrib.intersection)));
        float3 gammaK3 = normalize(cross((c - attrib.intersection), (a - attrib.intersection)));
        float3 tg3 = thetaK3 * gammaK3;

        float3 Phi = (tg + tg1 + tg2 + tg3) / 2.0f;

        result += brdf * qlights[i].intensity * dot(Phi, attrib.normal);
    }

    // Compute the final radiance
    payload.radiance = result * payload.throughput;

    // Calculate reflection
    if (length(mv.specular) > 0)
    {
        // Set origin and dir for tracing the reflection ray
        payload.origin = attrib.intersection;
        payload.dir = reflect(-attrib.wo, attrib.normal); // mirror reflection

        payload.depth++;
        payload.throughput *= mv.specular;
    }
    else
    {
        payload.done = true;
    }
}