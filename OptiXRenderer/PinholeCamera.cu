#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera varaibles here
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, up, , );
rtDeclareVariable(float3, center, , );
rtDeclareVariable(float1, fovx, , );
rtDeclareVariable(float1, fovy, , );
rtDeclareVariable(float1, width, , );
rtDeclareVariable(float1, height, , );

RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f);

    float3 origin = eye; 
    
    float epsilon = 0.001f; 

    float3 w = normalize(eye - center);
    float3 u = normalize(cross(up, w));
    float3 v = cross(w, u);

    float alpha = tan(fovx.x / 2.f) * ((((float)launchIndex.x + 0.5) - (width.x / 2.f)) / (width.x / 2.f));
    float beta = tan(fovy.x / 2.f) * ((((float)launchIndex.y + 0.5) - (height.x / 2.f)) / (height.x / 2.f));
    float3 dir = normalize(alpha * u + beta * v - w);

    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
    Payload payload;
    rtTrace(root, ray, payload);

    // Write the result
    resultBuffer[launchIndex] = payload.radiance;
}