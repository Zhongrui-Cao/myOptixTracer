#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera varaibles here
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, up, , );
rtDeclareVariable(float3, center, , );
rtDeclareVariable(float1, fovx, , );
rtDeclareVariable(float1, fovy, , );
rtDeclareVariable(float1, width, , );
rtDeclareVariable(float1, height, , );

RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f);
    size_t2 resultSize = resultBuffer.size();
    unsigned int index = launchIndex.x * resultSize.y + launchIndex.y;
    unsigned int seed = tea<16>(index * frameID.x, 0);

    float3 origin = eye; 

    float3 w = normalize(eye - center);
    float3 u = normalize(cross(up, w));
    float3 v = normalize(cross(w, u));

    float2 tanHFov = make_float2(tan(fovx.x / 2.f), tan(fovy.x / 2.f));
    float2 hSize = optix::make_float2(width.x / 2.f, height.x / 2.f);
    float2 xy = make_float2(launchIndex);
    xy.x += frameID.x == 1 ? 0.5f : rnd(seed);
    xy.y += frameID.x == 1 ? 0.5f : rnd(seed);

    float2 ab = tanHFov * (xy - hSize) / hSize;
    float3 dir = normalize(ab.x * u + ab.y * v - w); // ray direction

    //float alpha = tan(fovx.x / 2.f) * ((((float)launchIndex.x + 0.5) - (width.x / 2.f)) / (width.x / 2.f));
    //float beta = tan(fovy.x / 2.f) * ((((float)launchIndex.y + 0.5) - (height.x / 2.f)) / (height.x / 2.f));
    //float3 dir = normalize(alpha * u + beta * v - w);

    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    Payload payload;
    payload.radiance = make_float3(0.f);
    payload.throughput = make_float3(1.0f);
    payload.depth = 0;
    payload.done = false;
    int i = 0;

    do
    {
        payload.seed = tea<16>(index * frameID.x, i++);

        // Trace a ray
        Ray ray = make_Ray(origin, dir, 0, 0.001f, RT_DEFAULT_MAX);
        rtTrace(root, ray, payload);

        // Accumulate radiance
        result += payload.radiance;
        payload.radiance = make_float3(0.f);

        // Prepare to shoot next ray
        origin = payload.origin;
        dir = payload.dir;
    } while (!payload.done && payload.depth != 5);

    if (frameID.x == 1) 
        resultBuffer[launchIndex] = result;
    else
    {
        float u = 1.0f / (float)frameID.x;
        float3 oldResult = resultBuffer[launchIndex];
        resultBuffer[launchIndex] = lerp(oldResult, result, u);
    }
}