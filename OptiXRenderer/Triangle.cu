#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;

    // TODO: implement triangle intersection test here
    float3 normal = normalize(cross((tri.v1 - tri.v0), (tri.v2 - tri.v0)));
    float normray = dot(ray.direction, normal);

    //check if parallel
    if (fabs(normray) < ray.tmin)
        return;

    float An = dot(tri.v0, normal);
    float P0n = dot(ray.origin, normal);

    // intersect distance
    t = (An - P0n) / normray;
    // intersect point
    float3 p = ray.origin + t * ray.direction;

    // test if inside triangle using lamda
    float lamda0;
    float3 edge0 = tri.v1 - tri.v0;
    float3 vp0 = p - tri.v0;
    lamda0 = dot(normal, cross(edge0, vp0));

    float3 edge1 = tri.v2 - tri.v1;
    float3 vp1 = p - tri.v1;
    float lamda1 = dot(normal, cross(edge1, vp1));

    float3 edge2 = tri.v0 - tri.v2;
    float3 vp2 = p - tri.v2;
    float lamda2 = dot(normal, cross(edge2, vp2));

    if (lamda0 < 0 || lamda1 < 0 || lamda2 < 0)
        return;

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib.phongmat = tri.phongmat;
        attrib.intersection = p;
        attrib.wo = -ray.direction;
        attrib.normal = dot(tri.normal, -ray.direction) > 0 ? tri.normal : -tri.normal;
        attrib.isQuadLight = tri.isLight;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = fminf(fminf(tri.v0.x, tri.v1.x), tri.v2.x);
    result[1] = fminf(fminf(tri.v0.y, tri.v1.y), tri.v2.y);
    result[2] = fminf(fminf(tri.v0.z, tri.v1.z), tri.v2.z);
    result[3] = fmaxf(fmaxf(tri.v0.x, tri.v1.x), tri.v2.x);
    result[4] = fmaxf(fmaxf(tri.v0.y, tri.v1.y), tri.v2.y);
    result[5] = fmaxf(fmaxf(tri.v0.z, tri.v1.z), tri.v2.z);
}