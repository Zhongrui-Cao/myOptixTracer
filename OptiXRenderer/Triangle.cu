#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;

    // TODO: implement triangle intersection test here
    float3 normal = normalize(cross((tri.v1 - tri.v0), (tri.v2 - tri.v0)));
    float normray = dot(ray.direction, normal);

    //check if parallel
    if (fabs(normray) < ray.tmin)
        return;

    float An = dot(tri.v0, normal);
    float P0n = dot(ray.origin + ray.tmin * ray.direction, normal);

    // intersect distance
    t = (An - P0n) / normray;
    // intersect point
    float3 p = ray.origin + t * ray.direction;

    // test if inside triangle using lamda
    float lamda0;
    float3 edge0 = tri.v1 - tri.v0;
    float3 vp0 = p - tri.v0;
    lamda0 = dot(normal, cross(edge0, vp0));

    float3 edge1 = tri.v2 - tri.v1;
    float3 vp1 = p - tri.v1;
    float lamda1 = dot(normal, cross(edge1, vp1));

    float3 edge2 = tri.v0 - tri.v2;
    float3 vp2 = p - tri.v2;
    float lamda2 = dot(normal, cross(edge2, vp2));

    if (lamda0 < 0 || lamda1 < 0 || lamda2 < 0)
        return;

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib = tri.attribute;
        // TODO: assign attribute variables here

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}